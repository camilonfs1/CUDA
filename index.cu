#include "hip/hip_runtime.h"
// %%writefile cuda_filtro.cu
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <unistd.h>
#include <pthread.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "sod/sod.h"

#define MAX_H 4320
#define MAX_W 8192
#define MAX_INTERVAL 2048
#define INIT_KERNEL float kernel[3][3] = {{-2,-1,0}, {-1, 1,1}, {0,1,2}};
//#define INIT_KERNEL float kernel[3][3] = {{0,1,0}, {1, 4,1}, {0,1,0}};
//#define INIT_KERNEL float kernel[3][3] = {{0,1,0}, {1, 4,1}, {0,1,0}};


char* INIMAGE;
char* OUTIMAGE;
int ARG;
int THREADSNUM;
int BLOCKS;
int THREADSGPU;
int INTERVAL[MAX_INTERVAL][2];


__global__ 
void filter(int d_interval[MAX_INTERVAL][2], float (*d_board)[MAX_W], float (*d_output)[MAX_W], int *d_W, int *d_blocks) {
    int ID = blockIdx.x * blockDim.x + threadIdx.x;
    if (ID < (*d_blocks)) {        
        INIT_KERNEL;        
        int from = d_interval[ID][0];//Interval
        int to = d_interval[ID][1];
        for(int y = from; y <= to; ++y) {
            for(int x = 1; x < (*d_W)-1; ++x) {
                float sum = 0.0;
                for(int ky = -1; ky <= 1; ++ky) {
                    for(int kx = -1; kx <= 1; ++kx) {
                        float val = d_board[x+kx][y+ky];
                        sum += kernel[ky+1][kx+1] * val; //Filter
                    }
                }
                d_output[x][y] = abs(sum);
            }
        }
    }
}

double Time() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

int main(int argc, char *argv[]) {

    //Paths
    INIMAGE = argv[1];
    OUTIMAGE = argv[2];
    ARG = atof(argv[3]);
    //Size
    BLOCKS = atoi(argv[4]);
    THREADSGPU = atoi(argv[5]);
    THREADSNUM = BLOCKS * THREADSGPU;  
    sod_img imgIn;
    sod_img imgOut; 

    //Memory
    imgIn = sod_img_load_from_file(INIMAGE, SOD_IMG_COLOR);
    imgOut = sod_img_load_from_file(INIMAGE, SOD_IMG_COLOR);
    
    if (imgIn.data == 0) { // Image validation       
        printf("Image not found %s\n", INIMAGE);
        return 0;
    }

    //Intervals
    int factor = imgIn.h / THREADSNUM;
    int last = 1;
    for(int i = 0; i < THREADSNUM; ++i) {
        INTERVAL[i][0] = last;
        if(i != (THREADSNUM-1)) {
            INTERVAL[i][1] = last + factor-1;
        } else {
            INTERVAL[i][1] = imgIn.h - 1;
        }
        last = INTERVAL[i][1] + 1;
    }

    //printf("width=%d height=%d\n", imgIn.w, imgIn.h);

    // memory
    // host board
    float (*board)[MAX_W] = (float (*)[MAX_W]) malloc(MAX_H*MAX_W*sizeof(float));

    for(int y = 0; y <= imgIn.h; ++y) {
        for(int x = 0; x < imgIn.w; ++x) {
            float val = sod_img_get_pixel(imgIn, x, y, 0); // RED
            board[x][y] = val;
        }
    }

    // device board
    float (*d_board)[MAX_W];
    hipMalloc(&d_board, MAX_H*MAX_W*sizeof(float));
    hipMemcpy(d_board, board, MAX_H*MAX_W*sizeof(float), hipMemcpyHostToDevice);

    // Device output
    float (*d_output)[MAX_W];
    hipMalloc(&d_output, MAX_H*MAX_W*sizeof(float));

    // Device Interval
    int (*d_intervalo)[2];
    hipMalloc(&d_intervalo, MAX_INTERVAL*2*sizeof(int));
    hipMemcpy(d_intervalo, INTERVAL, MAX_INTERVAL*2*sizeof(int), hipMemcpyHostToDevice);

    // Device W
    int *d_W;
    int *tmp_W;
    int tmp = imgIn.w;
    tmp_W = &tmp;
    hipMalloc(&d_W, sizeof(int));
    hipMemcpy(d_W, tmp_W, sizeof(int), hipMemcpyHostToDevice);

    int *d_blocks;
    int *threads;
    threads = &THREADSNUM;
    hipMalloc(&d_blocks, sizeof(int));
    hipMemcpy(d_blocks, threads, sizeof(int), hipMemcpyHostToDevice);

    double start = Time(); // Get start time
    filter<<<BLOCKS, THREADSGPU>>>(d_intervalo, d_board, d_output, d_W, d_blocks); // Run filter
    hipDeviceSynchronize();
    double stop = Time();

    hipMemcpy(board, d_output, MAX_H*MAX_W*sizeof(float), hipMemcpyDeviceToHost);

    // To black and white
    for(int y = 0; y <= imgIn.h; ++y) {
        for(int x = 0; x < imgIn.w; ++x) {
            float val = board[x][y];
            sod_img_set_pixel(imgOut, x, y, 0, abs(val));
            sod_img_set_pixel(imgOut, x, y, 1, abs(val));
            sod_img_set_pixel(imgOut, x, y, 2, abs(val));
        }
    }        
    sod_img_save_as_png(imgOut, OUTIMAGE);// Save image

    // Free memory
    sod_free_image(imgIn);
    sod_free_image(imgOut);

    double time_elapsed = stop - start;

    // Time log
    printf("\nTime: %.8f blocks:%d, threads:%d\n", time_elapsed, BLOCKS, THREADSGPU);
    fflush(stdout);

    return 0;
}